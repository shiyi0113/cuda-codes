#include <iostream>
#include <hip/hip_runtime.h>


#define A(i,j) a[(i)*N + (j)]
#define B(i,j) b[(i)*N + (j)]
void random_matirx(const int M,const int N,float *a){
    for(int i=0;i<M;i++){
        for(int j=0;j<N;j++){
#if 1
            A(i,j) = 2.0*(float)drand48()-1.0;
#else
            A(i,j) = (j-i)%3;
#endif
        }
    }
}

float compare_matrices(const int M,const int N,float *a,float *b){
    float max_diff = 0.0f;
    float diff;
    int printed = 0;
    for(int i=0;i<M;i++){
        for(int j=0;j<N;j++){
            diff = abs(A(i,j)-B(i,j));
            max_diff = (diff>max_diff?diff:max_diff);
            if(0 == printed){
                if(max_diff>0.5f){
                    printf("\n error:i %d j %d diff %f got %f expect %f ",i,j,max_diff,A(i,j),B(i,j));
                    printed = 1;
                }
            }
        }
    }
    return max_diff;
}

void cpu_sgemm(float* A,float* B,float* C,const int M,const int N,const int K){
    for(int m=0;m<M;m++){
        for(int n=0;n<N;n++){
            float sum = 0;
            for(int k=0;k<K;k++){
                sum += A[m*K + k]*B[k*N + n];
            }
            C[m*N+n] = sum;
        }
    }
}

template<unsigned int BLOCK,unsigned int COARSEFACTOR>
__global__ void sgemm_threadCoarsening(float* d_A,float* d_B,float* d_C,const int M,const int N,const int K){
    constexpr int BLOCKNUM = BLOCK * COARSEFACTOR;
    __shared__ float s_A[BLOCKNUM][BLOCKNUM];
    __shared__ float s_B[BLOCKNUM][BLOCKNUM];
    float r_C[COARSEFACTOR][COARSEFACTOR];
    for(int i = 0;i < COARSEFACTOR;i++){
        for(int j = 0;j < COARSEFACTOR;j++){
            r_C[i][j] = 0.0f;
        }
    }
    float* A_begin = d_A + blockIdx.x * BLOCKNUM * K;
    float* B_begin = d_B + blockIdx.y * BLOCKNUM;
    const int C_m = blockIdx.x * BLOCKNUM + threadIdx.x;
    const int C_n = blockIdx.y * BLOCKNUM + threadIdx.y;

    for(size_t step = 0;step < (K + BLOCKNUM - 1)/BLOCKNUM;step++){
        for(int i = 0;i < COARSEFACTOR;i++){
            for(int j = 0;j < COARSEFACTOR;j++){
                int tx = threadIdx.x + i * BLOCK;
                int ty = threadIdx.y + j * BLOCK;
                s_A[tx][ty] = A_begin[tx * K + ty + step*BLOCKNUM];
                s_B[tx][ty] = B_begin[(tx + step*BLOCKNUM)*N + ty];
            }
        }
        __syncthreads();
        for(int i = 0;i < COARSEFACTOR;i++){
            for(int j = 0;j < COARSEFACTOR;j++){
                int tx = threadIdx.x + i * BLOCKNUM/2;
                int ty = threadIdx.y + j * BLOCKNUM/2;
                for(int k = 0;k < BLOCKNUM;k++)
                    r_C[i][j] += s_A[tx][k] * s_B[k][ty];
            }
        }
        __syncthreads();
    }
    for(int i = 0;i < COARSEFACTOR;i++){
        for(int j = 0;j < COARSEFACTOR;j++){
            d_C[(C_m + i * BLOCK) * N + C_n + j * BLOCK] = r_C[i][j];
        }
    }
}

int main(){
    const int M = 512;
    const int N = 512;
    const int K = 512;
    const size_t mem_size_A = M * K *sizeof(float);
    const size_t mem_size_B = K * N *sizeof(float);
    const size_t mem_size_C = M * N *sizeof(float);

    float* h_matrix_A = (float*)malloc(mem_size_A);
    float* h_matrix_B = (float*)malloc(mem_size_B);
    float* h_matrix_C = (float*)malloc(mem_size_C);
    float* h_matrix_C_cpu = (float*)malloc(mem_size_C);

    random_matirx(M,K,h_matrix_A);
    random_matirx(K,N,h_matrix_B);
    memset(h_matrix_C,0,mem_size_C);
    memset(h_matrix_C_cpu,0,mem_size_C);

    // cpu_calc
    cpu_sgemm(h_matrix_A,h_matrix_B,h_matrix_C_cpu,M,N,K);

    // gpu_calc
    float *d_matrix_A,*d_matrix_B,*d_matrix_C;
    hipMalloc((void**)&d_matrix_A,mem_size_A);
    hipMalloc((void**)&d_matrix_B,mem_size_B);
    hipMalloc((void**)&d_matrix_C,mem_size_C);
    hipMemcpy(d_matrix_A,h_matrix_A,mem_size_A,hipMemcpyHostToDevice);
    hipMemcpy(d_matrix_B,h_matrix_B,mem_size_B,hipMemcpyHostToDevice);

    const int BLOCK = 16;
    const int COARSEFACTOR = 2;
    dim3 Grid((M+BLOCK-1)/BLOCK/COARSEFACTOR,(N+BLOCK-1)/BLOCK/COARSEFACTOR);
    dim3 Block(BLOCK,BLOCK);
    sgemm_threadCoarsening<BLOCK,COARSEFACTOR><<<Grid,Block>>>(d_matrix_A,d_matrix_B,d_matrix_C,M,N,K);
    hipMemcpy(h_matrix_C,d_matrix_C,mem_size_C,hipMemcpyDeviceToHost);
    hipError_t err = hipGetLastError();
    if(err !=hipSuccess){
        std::cout<<"cuda Error: "<< hipGetErrorString(err)<<std::endl;
    }
    // check
    float diff = compare_matrices(M,N,h_matrix_C,h_matrix_C_cpu);
    if(diff > 0.5f){
        printf("diff too big !\n");
        exit(-1);
    }else{
        printf("right!\n");
    }
    // free
    free(h_matrix_A);
    free(h_matrix_B);
    free(h_matrix_C);
    free(h_matrix_C_cpu);
    hipFree(d_matrix_A);
    hipFree(d_matrix_B);
    hipFree(d_matrix_C);
}